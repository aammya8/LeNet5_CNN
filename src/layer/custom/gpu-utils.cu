#include "hip/hip_runtime.h"
#include "gpu-utils.h"

__global__ void do_not_remove_this_kernel() {
    int tx = threadIdx.x;
    tx = tx + 1;
}

__global__ void prefn_marker_kernel() {
    int tx = threadIdx.x;
    tx = tx + 1;
}

__host__ void GPU_Utils::insert_post_barrier_kernel() {
    
    dim3 GridDim(1,1,1);
    dim3 BlockDim(1,1,1);
    do_not_remove_this_kernel<<<GridDim, BlockDim>>>();
    hipDeviceSynchronize();
}

__host__ void GPU_Utils::insert_pre_barrier_kernel() {

    int* devicePtr;
    int x = 1;

    hipMalloc((void**) &devicePtr, sizeof(int));
    hipMemcpy(devicePtr, &x, sizeof(int), hipMemcpyHostToDevice);

    dim3 GridDim(1,1,1);
    dim3 BlockDim(1,1,1);
    prefn_marker_kernel<<<GridDim, BlockDim>>>();
    hipFree(devicePtr);
    hipDeviceSynchronize();
}